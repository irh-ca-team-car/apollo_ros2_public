#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright 2018 The Apollo Authors. All Rights Reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the License);
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an AS IS BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *****************************************************************************/
#include <algorithm>
#include <functional>
#include <map>
#include <memory>
#include <utility>
#include <vector>
#include "boost/iterator/counting_iterator.hpp"
#include "thrust/functional.h"
#include "thrust/sort.h"

#include "modules/perception/base/object_types.h"
#include "modules/perception/camera/lib/obstacle/detector/yolov4/object_maintainer.h"
#include "modules/perception/camera/lib/obstacle/detector/yolov4/region_output.h"

namespace apollo {
namespace perception {
namespace camera {

__host__ __device__ float sigmoid_gpu(float x) { return 1.0 / (1.0 + exp(-x)); }

__host__ __device__ float bbox_size_gpu(const float *bbox,
                                        const bool normalized) {
  if (bbox[2] <= bbox[0] || bbox[3] <= bbox[1]) {
    // If bbox is invalid (e.g. xmax < xmin or ymax < ymin), return 0.
    return 0.f;  // NOLINT
  } else {
    const float width = bbox[2] - bbox[0];
    const float height = bbox[3] - bbox[1];
    if (normalized) {
      return width * height;
    } else {
      // If bbox is not within range [0, 1].
      return (width + 1) * (height + 1);
    }
  }
}

__host__ __device__ float jaccard_overlap_gpu(const float *bbox1,
                                              const float *bbox2) {
  if (bbox2[0] > bbox1[2] || bbox2[2] < bbox1[0] || bbox2[1] > bbox1[3] ||
      bbox2[3] < bbox1[1]) {
    return float(0.);  // NOLINT
  } else {
    const float inter_xmin = max(bbox1[0], bbox2[0]);
    const float inter_ymin = max(bbox1[1], bbox2[1]);
    const float inter_xmax = min(bbox1[2], bbox2[2]);
    const float inter_ymax = min(bbox1[3], bbox2[3]);

    const float inter_width = inter_xmax - inter_xmin;
    const float inter_height = inter_ymax - inter_ymin;
    const float inter_size = inter_width * inter_height;

    const float bbox1_size = bbox_size_gpu(bbox1, true);
    const float bbox2_size = bbox_size_gpu(bbox2, true);

    return inter_size / (bbox1_size + bbox2_size - inter_size);
  }
}

__global__ void get_object_kernel(
    int n, const float *loc_data, const float *obj_data, const float *cls_data,
    const float *ori_data, const float *dim_data, const float *lof_data,
    const float *lor_data, const float *area_id_data,
    const float *visible_ratio_data, const float *cut_off_ratio_data,
    const float *brvis_data, const float *brswt_data, const float *ltvis_data,
    const float *ltswt_data, const float *rtvis_data, const float *rtswt_data,
    const float *anchor_data, const float *expand_data, int width, int height,
    int num_anchors, int num_classes, float confidence_threshold,
    float light_vis_conf_threshold, float light_swt_conf_threshold,
    bool with_box3d, bool with_frbox, bool with_lights, bool with_ratios,
    bool multi_scale, int num_areas, float *res_box_data, float *res_cls_data,
    int res_cls_offset, int all_scales_num_candidates) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n);
       i += blockDim.x * gridDim.x) {
    int box_block = kBoxBlockSize;

    int idx = i;
    int c = idx % num_anchors;
    idx = idx / num_anchors;
    int w = idx % width;
    idx = idx / width;
    int h = idx;
    int loc_index = (h * width + w) * num_anchors + c;
    int offset_loc = loc_index * 4;
    int offset_cls = loc_index * num_classes;
    float scale = obj_data[loc_index];
    float cx = (w + sigmoid_gpu(loc_data[offset_loc + 0])) / width;
    float cy = (h + sigmoid_gpu(loc_data[offset_loc + 1])) / height;
    float hw =
        exp(max(minExpPower, min(loc_data[offset_loc + 2], maxExpPower))) *
        anchor_data[2 * c] / width * 0.5;
    float hh =
        exp(max(minExpPower, min(loc_data[offset_loc + 3], maxExpPower))) *
        anchor_data[2 * c + 1] / height * 0.5;

    float max_prob = 0.f;
    int max_index = 0;
    for (int k = 0; k < num_classes; ++k) {
      float prob = cls_data[offset_cls + k] * scale;
      res_cls_data[k * all_scales_num_candidates + res_cls_offset + i] = prob;
      if (prob > max_prob) {
        max_prob = prob;
        max_index = k;
      }
    }
    res_cls_data[num_classes * all_scales_num_candidates + res_cls_offset + i] =
        max_prob;

    auto &&dst_ptr = res_box_data + i * box_block;
    hw += expand_data[max_index];
    dst_ptr[0] = cx - hw;
    dst_ptr[1] = cy - hh;
    dst_ptr[2] = cx + hw;
    dst_ptr[3] = cy + hh;

    if (with_box3d) {
      int offset_ori = loc_index * 2;
      dst_ptr[4] = atan2(ori_data[offset_ori + 1], ori_data[offset_ori]);

      int offset_dim = loc_index * 3;
      if (multi_scale) {
        offset_dim = loc_index * num_classes * 3 + max_index * 3;
      }
      dst_ptr[5] = dim_data[offset_dim + 0];
      dst_ptr[6] = dim_data[offset_dim + 1];
      dst_ptr[7] = dim_data[offset_dim + 2];
    }

    if (with_frbox) {
      {
        int offset_lof = loc_index * 4;
        auto &&src_ptr = lof_data + offset_lof;
        auto sb_x = src_ptr[0] * hw * 2 + cx;
        auto sb_y = src_ptr[1] * hh * 2 + cy;
        auto sb_hw = exp(src_ptr[2]) * hw;
        auto sb_hh = exp(src_ptr[3]) * hh;
        dst_ptr[8] = sb_x - sb_hw;
        dst_ptr[9] = sb_y - sb_hh;
        dst_ptr[10] = sb_x + sb_hw;
        dst_ptr[11] = sb_y + sb_hh;
      }

      {
        int offset_lor = loc_index * 4;
        auto &&src_ptr = lor_data + offset_lor;
        auto sb_x = src_ptr[0] * hw * 2 + cx;
        auto sb_y = src_ptr[1] * hh * 2 + cy;
        auto sb_hw = exp(src_ptr[2]) * hw;
        auto sb_hh = exp(src_ptr[3]) * hh;
        dst_ptr[12] = sb_x - sb_hw;
        dst_ptr[13] = sb_y - sb_hh;
        dst_ptr[14] = sb_x + sb_hw;
        dst_ptr[15] = sb_y + sb_hh;
      }
    }

    if (with_lights) {
      dst_ptr[16] = sigmoid_gpu(brvis_data[loc_index]);
      dst_ptr[17] = sigmoid_gpu(brswt_data[loc_index]);
      dst_ptr[18] = sigmoid_gpu(ltvis_data[loc_index]);
      dst_ptr[19] = sigmoid_gpu(ltswt_data[loc_index]);
      dst_ptr[20] = sigmoid_gpu(rtvis_data[loc_index]);
      dst_ptr[21] = sigmoid_gpu(rtswt_data[loc_index]);

      dst_ptr[16] = dst_ptr[16] > light_vis_conf_threshold ? dst_ptr[16] : 0;
      dst_ptr[18] = dst_ptr[18] > light_vis_conf_threshold ? dst_ptr[18] : 0;
      dst_ptr[20] = dst_ptr[20] > light_vis_conf_threshold ? dst_ptr[20] : 0;

      float swt_score = 0;
      swt_score = dst_ptr[16] * dst_ptr[17];
      dst_ptr[17] = swt_score > light_swt_conf_threshold ? swt_score : 0;

      swt_score = dst_ptr[18] * dst_ptr[19];
      dst_ptr[19] = swt_score > light_swt_conf_threshold ? swt_score : 0;

      swt_score = dst_ptr[20] * dst_ptr[21];
      dst_ptr[21] = swt_score > light_swt_conf_threshold ? swt_score : 0;
    }

    if (with_ratios) {
      // 0~3: cos2, left, visa, visb
      auto vis_pred = visible_ratio_data + loc_index * 4;
      auto vis_ptr = dst_ptr + 22;
      vis_ptr[0] = vis_ptr[1] = vis_ptr[2] = vis_ptr[3] = 0;
      const float hi_th = 0.75;
      const float lo_th = 1.f - hi_th;
      if (vis_pred[2] >= hi_th && vis_pred[3] >= hi_th) {  // 2 (1, 3)
        vis_ptr[0] = vis_pred[0];
        vis_ptr[1] = 1 - vis_pred[0];
      } else if (vis_pred[2] <= lo_th && vis_pred[3] >= hi_th) {  // 4 (3, 5)
        vis_ptr[2] = vis_pred[0];
        vis_ptr[1] = 1 - vis_pred[0];
      } else if (vis_pred[2] <= lo_th && vis_pred[3] <= lo_th) {  // 6 (5, 7)
        vis_ptr[2] = vis_pred[0];
        vis_ptr[3] = 1 - vis_pred[0];
      } else if (vis_pred[2] >= hi_th && vis_pred[3] <= lo_th) {  // 8 (7, 1)
        vis_ptr[0] = vis_pred[0];
        vis_ptr[3] = 1 - vis_pred[0];
      } else {
        vis_ptr[2] = vis_pred[0];
        if (vis_pred[1] > 0.5) {
          vis_ptr[1] = 1 - vis_pred[0];
        } else {
          vis_ptr[3] = 1 - vis_pred[0];
        }
      }

      int offset_cut = loc_index * 4;
      dst_ptr[26] = cut_off_ratio_data[offset_cut + 0];
      dst_ptr[27] = cut_off_ratio_data[offset_cut + 1];
      dst_ptr[28] = cut_off_ratio_data[offset_cut + 2];
      dst_ptr[29] = cut_off_ratio_data[offset_cut + 3];
    }

    if (num_areas > 0) {
      int offset_area_id = loc_index * num_areas;
      int max_area_id = 0;
      for (int area_id = 1; area_id < num_areas; ++area_id) {
        if (area_id_data[offset_area_id + area_id] >
            area_id_data[offset_area_id + max_area_id]) {
          max_area_id = area_id;
        }
      }
      dst_ptr[30] = max_area_id + 1;
      dst_ptr[31] = area_id_data[offset_area_id + max_area_id];
    }
  }
}

__global__ void get_rois_kernel(int num_bboxes, const float *loc_data,
                                const float *obj_data, const float *anchor_data,
                                int width, int height, int num_anchors,
                                float confidence_threshold, float *conf_data,
                                float *bbox_data) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < num_bboxes) {
    int offset_obj = idx;
    int offset_loc = idx * 4;

    int c = idx % num_anchors;
    idx /= num_anchors;
    int w = idx % width;
    idx /= width;
    int h = idx;

    float cx = (w + sigmoid_gpu(loc_data[offset_loc + 0])) / width;
    float cy = (h + sigmoid_gpu(loc_data[offset_loc + 1])) / height;
    float hw =
        exp(loc_data[offset_loc + 2]) * anchor_data[2 * c + 0] / width * 0.5;
    float hh =
        exp(loc_data[offset_loc + 3]) * anchor_data[2 * c + 1] / height * 0.5;

    const float &conf = obj_data[offset_obj];
    conf_data[offset_obj] = conf > confidence_threshold ? conf : 0;

    auto &&curr_bbox = bbox_data + offset_loc;
    curr_bbox[0] = cx - hw;
    curr_bbox[1] = cy - hh;
    curr_bbox[2] = cx + hw;
    curr_bbox[3] = cy + hh;
  }
}

__global__ void compute_overlapped_by_idx_kernel(
    const int nthreads, const float *bbox_data, const int bbox_step,
    const float overlap_threshold, const int *idx, const int num_idx,
    bool *overlapped_data) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < (nthreads);
       index += blockDim.x * gridDim.x) {
    const int j = index % num_idx;
    const int i = index / num_idx;
    if (i == j) {
      // Ignore same bbox.
      return;
    }
    // Compute overlap between i-th bbox and j-th bbox.
    const int start_loc_i = idx[i] * bbox_step;
    const int start_loc_j = idx[j] * bbox_step;
    const float overlap =
        jaccard_overlap_gpu(bbox_data + start_loc_i, bbox_data + start_loc_j);
    overlapped_data[index] = overlap > overlap_threshold;
  }
}

void compute_overlapped_by_idx_gpu(const int nthreads, const float *bbox_data,
                                   const int bbox_step,
                                   const float overlap_threshold,
                                   const int *idx, const int num_idx,
                                   bool *overlapped_data,
                                   const hipStream_t &stream) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  const int thread_size = 512;
  int block_size = (nthreads + thread_size - 1) / thread_size;
  compute_overlapped_by_idx_kernel<<<block_size, thread_size, 0, stream>>>(
      nthreads, bbox_data, bbox_step, overlap_threshold, idx, num_idx,
      overlapped_data);
}

void apply_nms_gpu(const float *bbox_data, const float *conf_data,
                   const std::vector<int> &origin_indices, const int bbox_step,
                   const float confidence_threshold, const int top_k,
                   const float nms_threshold, std::vector<int> *indices,
                   base::Blob<bool> *overlapped, base::Blob<int> *idx_sm,
                   const hipStream_t &stream) {
  // Keep part of detections whose scores are higher than confidence threshold.
  std::vector<int> idx;
  std::vector<float> confidences;
  for (auto i : origin_indices) {
    if (conf_data[i] > confidence_threshold) {
      idx.push_back(i);
      confidences.push_back(conf_data[i]);
    }
  }
  int num_remain = confidences.size();
  if (num_remain == 0) {
    return;
  }
  // Sort detections based on score.
  thrust::sort_by_key(&confidences[0], &confidences[0] + num_remain, &idx[0],
                      thrust::greater<float>());
  if (top_k > -1 && top_k < num_remain) {
    num_remain = top_k;
  }
  int *idx_data = (idx_sm->mutable_cpu_data());
  std::copy(idx.begin(), idx.begin() + num_remain, idx_data);

  overlapped->Reshape(std::vector<int>{num_remain, num_remain});
  bool *overlapped_data = (overlapped->mutable_gpu_data());

  compute_overlapped_by_idx_gpu(overlapped->count(), bbox_data, bbox_step,
                                nms_threshold, idx_sm->gpu_data(), num_remain,
                                overlapped_data, stream);

  // Do non-maximum suppression based on overlapped results.
  const bool *overlapped_results = (const bool *)overlapped->cpu_data();
  std::vector<int> selected_indices;

  apply_nms(overlapped_results, num_remain, &selected_indices);
  // Put back the selected information.
  for (size_t i = 0; i < selected_indices.size(); ++i) {
    indices->push_back(idx[selected_indices[i]]);
  }
}

void apply_nms(const bool *overlapped, const int num,
               std::vector<int> *indices) {
  std::vector<int> index_vec(boost::counting_iterator<int>(0),
                             boost::counting_iterator<int>(num));
  // Do nms.
  indices->clear();
  while (index_vec.size() != 0) {
    // Get the current highest score box.
    int best_idx = index_vec.front();
    indices->push_back(best_idx);
    // Erase the best box.
    index_vec.erase(index_vec.begin());

    for (std::vector<int>::iterator it = index_vec.begin();
         it != index_vec.end();) {
      int cur_idx = *it;

      // Remove it if necessary
      if (overlapped[best_idx * num + cur_idx]) {
        it = index_vec.erase(it);
      } else {
        ++it;
      }
    }
  }
}

const float *get_gpu_data(bool flag, const base::Blob<float> &blob) {
  return flag ? blob.gpu_data() : nullptr;
}

void get_intersect_bbox(const NormalizedBBox &bbox1,
                        const NormalizedBBox &bbox2,
                        NormalizedBBox *intersect_bbox) {
  if (bbox2.xmin > bbox1.xmax || bbox2.xmax < bbox1.xmin ||
      bbox2.ymin > bbox1.ymax || bbox2.ymax < bbox1.ymin) {
    // Return [0, 0, 0, 0] if there is no intersection.
    intersect_bbox->xmin = 0;
    intersect_bbox->ymin = 0;
    intersect_bbox->xmax = 0;
    intersect_bbox->ymax = 0;
  } else {
    intersect_bbox->xmin = std::max(bbox1.xmin, bbox2.xmin);
    intersect_bbox->ymin = std::max(bbox1.ymin, bbox2.ymin);
    intersect_bbox->xmax = std::min(bbox1.xmax, bbox2.xmax);
    intersect_bbox->ymax = std::min(bbox1.ymax, bbox2.ymax);
  }
}

float get_bbox_size(const NormalizedBBox &bbox) {
  if (bbox.xmax < bbox.xmin || bbox.ymax < bbox.ymin) {
    // If bbox is invalid (e.g. xmax < xmin or ymax < ymin), return 0.
    return 0;
  } else {
    if (bbox.size >= 0) {
      return bbox.size;
    } else {
      float width = bbox.xmax - bbox.xmin;
      float height = bbox.ymax - bbox.ymin;
      return width * height;
    }
  }
}

float get_jaccard_overlap(const NormalizedBBox &bbox1,
                          const NormalizedBBox &bbox2) {
  NormalizedBBox intersect_bbox;
  get_intersect_bbox(bbox1, bbox2, &intersect_bbox);
  float intersect_width = 0.f;
  float intersect_height = 0.f;
  intersect_width = intersect_bbox.xmax - intersect_bbox.xmin;
  intersect_height = intersect_bbox.ymax - intersect_bbox.ymin;

  if (intersect_width > 0 && intersect_height > 0) {
    float intersect_size = intersect_width * intersect_height;
    float bbox1_size = get_bbox_size(bbox1);
    float bbox2_size = get_bbox_size(bbox2);
    return intersect_size / (bbox1_size + bbox2_size - intersect_size);
  } else {
    return 0.;
  }
}

void get_max_score_index(const std::vector<float> &scores,
                         const float threshold, const int top_k,
                         std::vector<std::pair<float, int>> *score_index_vec) {
  // Generate index score pairs.
  for (size_t i = 0; i < scores.size(); ++i) {
    if (scores[i] > threshold) {
      score_index_vec->push_back(std::make_pair(scores[i], i));
    }
  }

  // Sort the score pair according to the scores in descending order
  std::stable_sort(score_index_vec->begin(), score_index_vec->end(),
                   sort_score_pair_descend<int>);

  // Keep top_k scores if needed.
  if (top_k > -1 && top_k < static_cast<int>(score_index_vec->size())) {
    score_index_vec->resize(top_k);
  }
}

void apply_softnms_fast(const std::vector<NormalizedBBox> &bboxes,
                        std::vector<float> *scores, const float score_threshold,
                        const float nms_threshold, const int top_k,
                        std::vector<int> *indices, bool is_linear,
                        const float sigma) {
  // Sanity check.
  CHECK_EQ(bboxes.size(), scores->size())
      << "bboxes and scores have different size.";

  // Get top_k scores (with corresponding indices).
  std::vector<std::pair<float, int>> score_index_vec;
  get_max_score_index(*scores, score_threshold, top_k, &score_index_vec);

  // Do nms.
  indices->clear();
  while (score_index_vec.size() != 0) {
    auto best_it =
        max_element(std::begin(score_index_vec), std::end(score_index_vec));
    const int best_idx = (*best_it).second;
    score_index_vec.erase(best_it);
    const NormalizedBBox &best_bbox = bboxes[best_idx];
    indices->push_back(best_idx);
    for (std::vector<std::pair<float, int>>::iterator it =
             score_index_vec.begin();
         it != score_index_vec.end();) {
      int cur_idx = it->second;
      const NormalizedBBox &cur_bbox = bboxes[cur_idx];

      float cur_overlap = 0.;
      cur_overlap = get_jaccard_overlap(best_bbox, cur_bbox);
      if (is_linear) {
        (*scores)[cur_idx] *= (1.0 - cur_overlap);
      } else {
        (*scores)[cur_idx] *= exp(-1.0 * pow(cur_overlap, 2) / sigma);
      }
      ++it;
    }
  }
}

void apply_boxvoting_fast(std::vector<NormalizedBBox> *bboxes,
                          std::vector<float> *scores,
                          const float conf_threshold, const float nms_threshold,
                          const float sigma, std::vector<int> *indices) {
  if (bboxes->size() == 0) {
    return;
  }
  indices->clear();
  for (size_t i = 0; i < bboxes->size(); ++i) {
    (*bboxes)[i].mask = false;
    if ((*scores)[i] > conf_threshold) {
      indices->push_back(i);
    }
  }
  for (size_t count = 0; count < indices->size(); ++count) {
    int max_box_idx = 0;

    for (size_t i = 1; i < indices->size(); ++i) {
      int idx = indices->at(i);
      if ((*bboxes)[idx].mask) {
        continue;
      }
      if ((*scores)[idx] > (*scores)[max_box_idx]) {
        max_box_idx = idx;
      }
    }

    NormalizedBBox &best_bbox = (*bboxes)[max_box_idx];
    best_bbox.score = (*scores)[max_box_idx];
    best_bbox.mask = true;
    float s_vt = (*scores)[max_box_idx];
    float x1_vt = best_bbox.xmin * s_vt;
    float x2_vt = best_bbox.xmax * s_vt;
    float y1_vt = best_bbox.ymin * s_vt;
    float y2_vt = best_bbox.ymax * s_vt;
    for (size_t i = 0; i < indices->size(); ++i) {
      int sub_it = indices->at(i);
      if ((*bboxes)[sub_it].mask) {
        continue;
      }
      float cur_overlap = 0.;
      cur_overlap = get_jaccard_overlap(best_bbox, (*bboxes)[sub_it]);
      if (sigma == 0) {
        (*bboxes)[sub_it].mask = true;
      } else {
        (*scores)[sub_it] *= exp(-1.0 * pow(cur_overlap, 2) / sigma);
      }
      (*bboxes)[sub_it].score = (*scores)[sub_it];

      // Remove it if necessary
      if (cur_overlap > nms_threshold) {
        float s_vt_cur = (*bboxes)[sub_it].score;
        s_vt += s_vt_cur;
        x1_vt += (*bboxes)[sub_it].xmin * s_vt_cur;
        x2_vt += (*bboxes)[sub_it].xmax * s_vt_cur;
        y1_vt += (*bboxes)[sub_it].ymin * s_vt_cur;
        y2_vt += (*bboxes)[sub_it].ymax * s_vt_cur;
      }
    }
    if (s_vt > 0.0001) {
      (*bboxes)[max_box_idx].xmin = x1_vt / s_vt;
      (*bboxes)[max_box_idx].xmax = x2_vt / s_vt;
      (*bboxes)[max_box_idx].ymin = y1_vt / s_vt;
      (*bboxes)[max_box_idx].ymax = y2_vt / s_vt;
    }
  }
}

void apply_nms_fast(const std::vector<NormalizedBBox> &bboxes,
                    const std::vector<float> &scores,
                    const float score_threshold, const float nms_threshold,
                    const float eta, const int top_k,
                    std::vector<int> *indices) {
  // Sanity check.
  CHECK_EQ(bboxes.size(), scores.size())
      << "bboxes and scores have different size.";

  // Get top_k scores (with corresponding indices).
  std::vector<std::pair<float, int>> score_index_vec;
  get_max_score_index(scores, score_threshold, top_k, &score_index_vec);

  // Do nms.
  float adaptive_threshold = nms_threshold;
  indices->clear();
  while (score_index_vec.size() != 0) {
    const int idx = score_index_vec.front().second;
    bool keep = true;
    for (size_t k = 0; k < indices->size(); ++k) {
      if (keep) {
        const int kept_idx = (*indices)[k];
        float overlap = get_jaccard_overlap(bboxes[idx], bboxes[kept_idx]);
        keep = overlap <= adaptive_threshold;
      } else {
        break;
      }
    }
    if (keep) {
      indices->push_back(idx);
    }
    score_index_vec.erase(score_index_vec.begin());
    if (keep && eta < 1 && adaptive_threshold > 0.5) {
      adaptive_threshold *= eta;
    }
  }
}

void filter_bbox(const MinDims &min_dims,
                 std::vector<base::ObjectPtr> *objects) {
  size_t valid_obj_idx = 0;
  size_t total_obj_idx = 0;
  while (total_obj_idx < objects->size()) {
    const auto &obj = (*objects)[total_obj_idx];
    if ((obj->camera_supplement.box.ymax - obj->camera_supplement.box.ymin) >=
            min_dims.min_2d_height &&
        (min_dims.min_3d_height <= 0 ||
         obj->size[2] >= min_dims.min_3d_height) &&
        (min_dims.min_3d_width <= 0 || obj->size[1] >= min_dims.min_3d_width) &&
        (min_dims.min_3d_length <= 0 ||
         obj->size[0] >= min_dims.min_3d_length)) {
      (*objects)[valid_obj_idx] = (*objects)[total_obj_idx];
      ++valid_obj_idx;
    }
    ++total_obj_idx;
  }
  objects->resize(valid_obj_idx);
}
void recover_bbox(int roi_w, int roi_h, int offset_y,
                  std::vector<base::ObjectPtr> *objects) {
  for (auto &obj : *objects) {
    float xmin = obj->camera_supplement.box.xmin;
    float ymin = obj->camera_supplement.box.ymin;
    float xmax = obj->camera_supplement.box.xmax;
    float ymax = obj->camera_supplement.box.ymax;
    int x = xmin * roi_w;
    int w = (xmax - xmin) * roi_w;
    int y = ymin * roi_h + offset_y;
    int h = (ymax - ymin) * roi_h;
    base::RectF rect_det(x, y, w, h);
    base::RectF rect_img(0, 0, roi_w, roi_h + offset_y);
    base::RectF rect = rect_det & rect_img;
    obj->camera_supplement.box = rect;

    double eps = 1e-2;

    // Truncation assignment based on bbox positions
    if ((ymin < eps) || (ymax >= 1.0 - eps)) {
      obj->camera_supplement.truncated_vertical = 0.5;
    } else {
      obj->camera_supplement.truncated_vertical = 0.0;
    }
    if ((xmin < eps) || (xmax >= 1.0 - eps)) {
      obj->camera_supplement.truncated_horizontal = 0.5;
    } else {
      obj->camera_supplement.truncated_horizontal = 0.0;
    }

    obj->camera_supplement.front_box.xmin *= roi_w;
    obj->camera_supplement.front_box.ymin *= roi_h;
    obj->camera_supplement.front_box.xmax *= roi_w;
    obj->camera_supplement.front_box.ymax *= roi_h;

    obj->camera_supplement.back_box.xmin *= roi_w;
    obj->camera_supplement.back_box.ymin *= roi_h;
    obj->camera_supplement.back_box.xmax *= roi_w;
    obj->camera_supplement.back_box.ymax *= roi_h;

    obj->camera_supplement.front_box.ymin += offset_y;
    obj->camera_supplement.front_box.ymax += offset_y;
    obj->camera_supplement.back_box.ymin += offset_y;
    obj->camera_supplement.back_box.ymax += offset_y;
  }
}

void fill_base(base::ObjectPtr obj, const float *bbox) {
  obj->camera_supplement.box.xmin = bbox[0];
  obj->camera_supplement.box.ymin = bbox[1];
  obj->camera_supplement.box.xmax = bbox[2];
  obj->camera_supplement.box.ymax = bbox[3];
}

void fill_bbox3d(bool with_box3d, base::ObjectPtr obj, const float *bbox) {
  if (with_box3d) {
    obj->camera_supplement.alpha = bbox[0];
    obj->size[2] = bbox[1];
    obj->size[1] = bbox[2];
    obj->size[0] = bbox[3];
  }
}

void fill_frbox(bool with_frbox, base::ObjectPtr obj, const float *bbox) {
  if (with_frbox) {
    obj->camera_supplement.front_box.xmin = bbox[0];
    obj->camera_supplement.front_box.ymin = bbox[1];
    obj->camera_supplement.front_box.xmax = bbox[2];
    obj->camera_supplement.front_box.ymax = bbox[3];

    obj->camera_supplement.back_box.xmin = bbox[4];
    obj->camera_supplement.back_box.ymin = bbox[5];
    obj->camera_supplement.back_box.xmax = bbox[6];
    obj->camera_supplement.back_box.ymax = bbox[7];
  }
}

void fill_lights(bool with_lights, base::ObjectPtr obj, const float *bbox) {
  if (with_lights) {
    obj->car_light.brake_visible = bbox[0];
    obj->car_light.brake_switch_on = bbox[1];
    obj->car_light.left_turn_visible = bbox[2];
    obj->car_light.left_turn_switch_on = bbox[3];
    obj->car_light.right_turn_visible = bbox[4];
    obj->car_light.right_turn_switch_on = bbox[5];
  }
}

void fill_ratios(bool with_ratios, base::ObjectPtr obj, const float *bbox) {
  if (with_ratios) {
    // visible ratios of face a/b/c/d
    obj->camera_supplement.visible_ratios[0] = bbox[0];
    obj->camera_supplement.visible_ratios[1] = bbox[1];
    obj->camera_supplement.visible_ratios[2] = bbox[2];
    obj->camera_supplement.visible_ratios[3] = bbox[3];

    // cut off on width and length (3D)
    obj->camera_supplement.cut_off_ratios[0] = bbox[4];
    obj->camera_supplement.cut_off_ratios[1] = bbox[5];
    // cut off on left and right side (2D)
    obj->camera_supplement.cut_off_ratios[2] = bbox[6];
    obj->camera_supplement.cut_off_ratios[3] = bbox[7];
  }
}

void fill_area_id(bool with_flag, base::ObjectPtr obj, const float *data) {
  if (with_flag) {
    obj->camera_supplement.area_id = static_cast<int>(data[0]);
    // obj->camera_supplement.area_id_prob = data[1];
  }
}

int get_area_id(float visible_ratios[4]) {
  int area_id = 0;
  int max_face = 0;
  for (int i = 1; i < 4; ++i) {
    if (visible_ratios[i] > visible_ratios[max_face]) {
      max_face = i;
    }
  }
  int left_face = (max_face + 1) % 4;
  int right_face = (max_face + 3) % 4;
  const float eps = 1e-3;
  float max_ratio = visible_ratios[max_face];
  float left_ratio = visible_ratios[left_face];
  float right_ratio = visible_ratios[right_face];
  memset(visible_ratios, 0, 4 * sizeof(visible_ratios[0]));
  if (left_ratio < eps && right_ratio < eps) {
    area_id = (max_face * 2 + 1);
    visible_ratios[max_face] = 1.f;
  } else if (left_ratio > right_ratio) {
    area_id = (max_face * 2 + 2);
    auto &&sum_ratio = left_ratio + max_ratio;
    visible_ratios[max_face] = max_ratio / sum_ratio;
    visible_ratios[left_face] = left_ratio / sum_ratio;
  } else {
    area_id = (max_face * 2);
    if (area_id == 0) {
      area_id = 8;
    }
    auto &&sum_ratio = right_ratio + max_ratio;
    visible_ratios[max_face] = max_ratio / sum_ratio;
    visible_ratios[right_face] = right_ratio / sum_ratio;
  }
  return area_id;
}

}  // namespace camera
}  // namespace perception
}  // namespace apollo
